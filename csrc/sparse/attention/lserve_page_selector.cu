#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#define CHECK_DEVICE(x) TORCH_CHECK(x.device().type() == torch::kCUDA, #x " must be on CUDA")
#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

#define DISPATCH_HALF_AND_BF16(TYPE, NAME, ...)                            \
  if (TYPE == at::ScalarType::Half) {                                      \
    using scalar_t = at::Half;                                             \
    __VA_ARGS__();                                                         \
  } else if (TYPE == at::ScalarType::BFloat16) {                           \
    using scalar_t = at::BFloat16;                                         \
    __VA_ARGS__();                                                         \
  } else {                                                                 \
    AT_ERROR(#NAME, " not implemented for type '", toString(TYPE), "'"); \
  }

#define DISPATCH_WITH_HEAD_DIM(HEAD_DIM_NAME, NAME, ...)                        \
    if (HEAD_DIM_NAME == 32) {                                           \
        constexpr int HEAD_DIM = 32;                                       \
        __VA_ARGS__();                                                     \
    } else if (HEAD_DIM_NAME == 64) {                                           \
        constexpr int HEAD_DIM = 64;                                       \
        __VA_ARGS__();                                                     \
    } else if (HEAD_DIM_NAME == 128) {                                          \
        constexpr int HEAD_DIM = 128;                                      \
        __VA_ARGS__();                                                     \
    } else if (HEAD_DIM_NAME == 256) {                                          \
        constexpr int HEAD_DIM = 256;                                      \
        __VA_ARGS__();                                                     \
    } else {                                                                \
        AT_ERROR(#NAME, " not implemented head dim"); \
    }

template<typename T>
struct SATypeConverter {
    using Type = T;
};

template<>
struct SATypeConverter<at::Half> {
    using Type = uint16_t;
};

template<>
struct SATypeConverter<at::BFloat16> {
    using Type = __hip_bfloat16;
};

/////////////////////////////// Q_VEC transfer type definition /////////////////////////
template <typename T, int Dh_MAX>
struct Q_VEC_TRANSFER
{
};

template <>
struct Q_VEC_TRANSFER<uint16_t, 32>
{
    using Type = uint32_t;
};

template <>
struct Q_VEC_TRANSFER<uint16_t, 64>
{
    using Type = uint32_t;
};

template <>
struct Q_VEC_TRANSFER<uint16_t, 128>
{
    using Type = uint2;
};

template <>
struct Q_VEC_TRANSFER<uint16_t, 256>
{
    using Type = uint4;
};

template <>
struct Q_VEC_TRANSFER<__hip_bfloat16, 32>
{
    using Type = uint32_t;
};

template <>
struct Q_VEC_TRANSFER<__hip_bfloat16, 64>
{
    using Type = uint32_t;
};

template <>
struct Q_VEC_TRANSFER<__hip_bfloat16, 128>
{
    using Type = uint2;
};

template <>
struct Q_VEC_TRANSFER<__hip_bfloat16, 256>
{
    using Type = uint4;
};
///////////////////////////////////////////////////////////////////

/////////////////////////////// CALC_VEC type definition /////////////////////////
template <typename T, int VEC_SIZE>
struct CALC_VEC
{
};

template <>
struct CALC_VEC<uint16_t, 2>
{
    using Type = uint32_t;
};

template <>
struct CALC_VEC<uint16_t, 4>
{
    using Type = uint2;
};

template <>
struct CALC_VEC<uint16_t, 8>
{
    using Type = uint4;
};

template <>
struct CALC_VEC<__hip_bfloat16, 2>
{
    using Type = uint32_t;
};

template <>
struct CALC_VEC<__hip_bfloat16, 4>
{
    using Type = uint2;
};

template <>
struct CALC_VEC<__hip_bfloat16, 8>
{
    using Type = uint4;
};
///////////////////////////////////////////////////////////////////

/////////////////////////////// QkDotMinMaxTypeConverter /////////////////////////
template <typename T, typename VecT>
struct QkDotMinMaxTypeConverter
{
};

template <>
struct QkDotMinMaxTypeConverter<uint16_t, uint4>
{
    using Type = half2;
};

template <>
struct QkDotMinMaxTypeConverter<__hip_bfloat16, uint4>
{
    using Type = __hip_bfloat162;
};
///////////////////////////////////////////////////////////////////

template <typename T>
inline __device__ __host__ constexpr unsigned threads_per_value(unsigned dh)
{
    return dh * sizeof(T) / 16;
}

template <int THREAD_PER_KEY>
inline __device__ float qk_hmma_dot_min_max(const half2* q, const half2* k_max, const half2* k_min)
{
    half2 acc_max = __hmul2(q[0], k_max[0]);
    half2 acc_min = __hmul2(q[0], k_min[0]);
    half2 acc = __hmax2(acc_max, acc_min);
#pragma unroll
    for (int ii = 1; ii < 4; ++ii)
    {
        acc_max = __hmul2(q[ii], k_max[ii]);
        acc_min = __hmul2(q[ii], k_min[ii]);
        acc = __hadd2(acc, __hmax2(acc_max, acc_min));
    }
    float qk_min_max = __half2float(__hadd(acc.x, acc.y));

#pragma unroll
    for (int mask = THREAD_PER_KEY / 2; mask >= 1; mask /= 2)
    {
        qk_min_max += __shfl_xor_sync(uint32_t(-1), qk_min_max, mask);
    }

    return qk_min_max;
}

template <int THREAD_PER_KEY>
inline __device__ float qk_hmma_dot_min_max(const __hip_bfloat162* q, const __hip_bfloat162* k_max, const __hip_bfloat162* k_min)
{
    __hip_bfloat162 acc_max = __hmul2(q[0], k_max[0]);
    __hip_bfloat162 acc_min = __hmul2(q[0], k_min[0]);
    __hip_bfloat162 acc = __hmax2(acc_max, acc_min);
#pragma unroll
    for (int ii = 1; ii < 4; ++ii)
    {
        acc_max = __hmul2(q[ii], k_max[ii]);
        acc_min = __hmul2(q[ii], k_min[ii]);
        acc = __hadd2(acc, __hmax2(acc_max, acc_min));
    }
    float qk_min_max = __bfloat162float(__hadd(acc.x, acc.y));

#pragma unroll
    for (int mask = THREAD_PER_KEY / 2; mask >= 1; mask /= 2)
    {
        qk_min_max += __shfl_xor_sync(uint32_t(-1), qk_min_max, mask);
    }

    return qk_min_max;
}

template <
    typename T, 
    int HEAD_DIM, 
    unsigned THREADS_PER_BLOCK = 256,
    unsigned THREAD_PER_KEY = threads_per_value<T>(HEAD_DIM),
    unsigned META_CACHE_BLOCKS_PER_THREAD_BLOCK = 64>  // 目前设置为 64
__global__ void lserve_page_selector_kernel(
    const T* q,
    const T* key_meta_cache,
    const int* block_table,
    const int* num_full_blocks,
    T* out,
    const int num_q_head,
    const int num_kv_head,
    const int max_block_size,

    const int64_t qstride0, const int64_t qstride1,
    const int64_t kmc_stride0, const int64_t kmc_stride1, const int64_t kmc_stride2,
    const int64_t out_stride0, const int64_t out_stride1
) {
    const auto tid = threadIdx.x;
    const auto batch_id = blockIdx.x;
    const auto qhead_id = blockIdx.y;
    const auto block_tile_id = blockIdx.z;
    const auto head_group_size = num_q_head / num_kv_head;
    const auto khead_id = qhead_id / head_group_size;

    // 0.0 判断现在要处理的 META_CACHE_BLOCKS_PER_THREAD_BLOCK 个 meta cache block 是否超过了 num_full_blocks
    // 如果超过则直接return
    const unsigned logical_meta_cache_blocks_id = META_CACHE_BLOCKS_PER_THREAD_BLOCK * block_tile_id;
    int num_max_blocks = num_full_blocks[batch_id];
    if (logical_meta_cache_blocks_id >= num_max_blocks) {
        return;
    }

    // 0.1 读取 block table 到 smem
    static_assert(THREADS_PER_BLOCK >= META_CACHE_BLOCKS_PER_THREAD_BLOCK);
    __shared__ __align__(sizeof(int)) int BLOCK_TABLE_FOR_THREAD_BLOCK[META_CACHE_BLOCKS_PER_THREAD_BLOCK];
    const int* block_table_gmem_ptr = block_table + batch_id * max_block_size + logical_meta_cache_blocks_id;
    if (logical_meta_cache_blocks_id + tid < num_max_blocks) {
        BLOCK_TABLE_FOR_THREAD_BLOCK[tid] = *(block_table_gmem_ptr + tid);
    } else if (tid < META_CACHE_BLOCKS_PER_THREAD_BLOCK) {
        // 设置为 0 号 block，以免后续引入复杂的边界检查
        BLOCK_TABLE_FOR_THREAD_BLOCK[tid] = 0;
    }
    // __syncthreads();

    using q_vec = typename Q_VEC_TRANSFER<T, HEAD_DIM>::Type;
    constexpr unsigned Q_VEC_SIZE = sizeof(q_vec) / sizeof(T);
    const auto qvec_offset = tid * Q_VEC_SIZE;
    const T* query_gmem_ptr = q + qstride0 * batch_id + qstride1 * qhead_id;
    __shared__ __align__(sizeof(q_vec)) T q_smem[HEAD_DIM];
    // 1. 装载 query 到 smem
    if (qvec_offset < HEAD_DIM) {
        *reinterpret_cast<q_vec*>(&(q_smem[qvec_offset])) = *reinterpret_cast<const q_vec*>(query_gmem_ptr + qvec_offset);
    }
    __syncthreads();

    // 2. 从 smem 装载到寄存器
    constexpr auto CALC_VEC_SIZE = 16u / sizeof(T);
    constexpr auto ELEMENTS_PER_CHUNK = THREAD_PER_KEY * CALC_VEC_SIZE;
    // 目前的检查下可以保证刚好可以处理一整个 HEAD_DIM
    static_assert(ELEMENTS_PER_CHUNK == HEAD_DIM);
    // 每个线程的 CALC_VEC 寄存器
    using cvec = typename CALC_VEC<T, CALC_VEC_SIZE>::Type;
    cvec calc_qvec;
    // 每个线程只需要独立处理 query 的 CALC_VEC_SIZE 大小即可
    const auto calc_qvec_offset = (tid % THREAD_PER_KEY) * CALC_VEC_SIZE;
    calc_qvec = *reinterpret_cast<cvec*>(&(q_smem[calc_qvec_offset]));

    // constexpr unsigned WARP_SIZE = 32;
    constexpr unsigned NUM_META_BLOCK_PER_THREAD_BLOCK = THREADS_PER_BLOCK / THREAD_PER_KEY;
    // constexpr unsigned NUM_META_BLOCK_PER_WARP = WARP_SIZE / THREAD_PER_KEY;
    static_assert(META_CACHE_BLOCKS_PER_THREAD_BLOCK % NUM_META_BLOCK_PER_THREAD_BLOCK == 0);
    constexpr auto ITERS = META_CACHE_BLOCKS_PER_THREAD_BLOCK / NUM_META_BLOCK_PER_THREAD_BLOCK;
    // 3. 从 gmem 中向量化 load 每个线程需要的数据
    cvec calc_max_kvec[ITERS];
    cvec calc_min_kvec[ITERS];
#pragma unroll
    for (int block_table_idx = tid / THREAD_PER_KEY, iter = 0; 
        block_table_idx < META_CACHE_BLOCKS_PER_THREAD_BLOCK; 
        block_table_idx += NUM_META_BLOCK_PER_THREAD_BLOCK, ++iter) 
    {
        int block_id = BLOCK_TABLE_FOR_THREAD_BLOCK[block_table_idx];
        const T* key_max_cache_gmem_ptr = key_meta_cache + block_id * kmc_stride0 + khead_id * kmc_stride0;
        const T* key_min_cache_gmem_ptr = key_max_cache_gmem_ptr + kmc_stride2;
        calc_max_kvec[iter] = *reinterpret_cast<const cvec*>(key_max_cache_gmem_ptr + calc_qvec_offset);
        calc_min_kvec[iter] = *reinterpret_cast<const cvec*>(key_min_cache_gmem_ptr + calc_qvec_offset);
    }
    __syncthreads();

    // 4. 每个线程计算局部向量点积，再reduce求和
    using qk_type = typename QkDotMinMaxTypeConverter<T, cvec>::Type;
    T* out_gmem_ptr = out + batch_id * out_stride0 + qhead_id * out_stride1;
    unsigned logical_block_id = logical_meta_cache_blocks_id + (tid / THREAD_PER_KEY);
#pragma unroll
    for (int iter = 0; iter < ITERS && logical_block_id < num_max_blocks;
         ++iter, logical_block_id += NUM_META_BLOCK_PER_THREAD_BLOCK) {
        qk_type* q = reinterpret_cast<qk_type*>(&(calc_qvec));
        qk_type* k_max = reinterpret_cast<qk_type*>(&(calc_max_kvec[iter]));
        qk_type* k_min = reinterpret_cast<qk_type*>(&(calc_min_kvec[iter]));
        float qk_min_max = qk_hmma_dot_min_max<THREAD_PER_KEY>(q, k_max, k_min);

        // 5. 结果写回 out
        if (calc_qvec_offset == 0) {
            if constexpr(std::is_same<T, uint16_t>::value) {
                *(out_gmem_ptr + logical_block_id) = __float2half(qk_min_max);
            } else {
                *(out_gmem_ptr + logical_block_id) = __float2bfloat16(qk_min_max);
            }
        }
    }
}

template<typename T, int HEAD_DIM>
void lserve_page_selector_kernel_launch(
    const T* q,
    const T* key_meta_cache,
    const int* block_table,
    const int* num_full_blocks,
    T* out,
    const int batch_size,
    const int num_q_head,
    const int num_kv_head,
    const int head_dim,
    const int max_block_size,

    const int64_t qstride0, const int64_t qstride1
) {
    // auto constexpr threads_per_value = threads_per_value<T>(HEAD_DIM);
    constexpr unsigned META_CACHE_BLOCKS_PER_THREAD_BLOCK = 64;
    const unsigned num_block_tiles = (max_block_size + META_CACHE_BLOCKS_PER_THREAD_BLOCK - 1) / META_CACHE_BLOCKS_PER_THREAD_BLOCK;
    const int64_t kmc_stride0 = ((int64_t)1) * num_kv_head * 2 * head_dim;
    const int64_t kmc_stride1 = ((int64_t)1) * 2 * head_dim;
    const int64_t kmc_stride2 = ((int64_t)1) * head_dim;
    const int64_t out_stride0 = ((int64_t)1) * num_q_head * max_block_size;
    const int64_t out_stride1 = ((int64_t)1) * max_block_size;
    dim3 grid(batch_size, num_q_head, num_block_tiles);
    dim3 block(256);
    lserve_page_selector_kernel<T, HEAD_DIM><<<grid, block>>>(
        q, key_meta_cache, block_table,
        num_full_blocks, out,
        num_q_head,
        num_kv_head,
        max_block_size,
        qstride0, qstride1, 
        kmc_stride0, kmc_stride1, kmc_stride2,
        out_stride0, out_stride1
    );
}

template<typename T>
void lserve_page_selector_head_dim_dispatcher(
    const T* q,
    const T* key_meta_cache,
    const int* block_table,
    const int* num_full_blocks,
    T* out,
    const int batch_size,
    const int num_q_head,
    const int num_kv_head,
    const int head_dim,
    const int max_block_size,

    const int64_t qstride0, const int64_t qstride1
) {
    DISPATCH_WITH_HEAD_DIM(head_dim, "lserve_page_selector_head_dim_dispatcher", [&] {
        lserve_page_selector_kernel_launch<T, HEAD_DIM>(
            q, key_meta_cache, block_table,
            num_full_blocks, out,
            batch_size, 
            num_q_head,
            num_kv_head,
            max_block_size,
            head_dim,
            qstride0, qstride1
        );
    });
}


void lserve_page_selector(
    const torch::Tensor& q,                   // [batch_size, num_q_head, head_dim]. Only for decode. bf16
    const torch::Tensor& key_meta_cache,      // [num_block, num_kv_head, 2, head_dim] bf16
    const torch::Tensor& block_table,         // [batch_size, max_block_size] int32
    const torch::Tensor& num_full_blocks,     // [batch_size, ] int32
    torch::Tensor& out                        // [batch_size, num_q_head, max_block_size] bf16
) {
    CHECK_DEVICE(q); CHECK_DEVICE(key_meta_cache); 
    CHECK_DEVICE(block_table); CHECK_DEVICE(num_full_blocks);
    CHECK_DEVICE(out);

    // q 不是连续的，需要通过 stride 寻址
    // key_meta_cache 每层连续
    CHECK_CONTIGUOUS(key_meta_cache); CHECK_CONTIGUOUS(block_table);
    CHECK_CONTIGUOUS(num_full_blocks); CHECK_CONTIGUOUS(out);
    TORCH_CHECK(q.stride(-1) == 1);

    TORCH_CHECK(block_table.dtype() == torch::kInt32);
    TORCH_CHECK(num_full_blocks.dtype() == torch::kInt32);

    int batch_size = q.size(0);
    int num_q_head = q.size(1);
    int head_dim = q.size(-1);
    int num_kv_head = key_meta_cache.size(0);
    TORCH_CHECK(num_q_head % num_kv_head == 0);

    int max_block_size = block_table.size(-1);
    TORCH_CHECK(out.size(-1) == max_block_size);
    // 1. 目前先支持 fp16/bf16 类型
    // 2. 每个线程处理 16B 数据, 8个fp16, 同时希望一个 query key 能够被一个 warp 处理, 那么 head_dim 最长只能是 32 * 8 = 256
    //    其次，希望一个 warp 能够处理整数个 qk， 所以需要是 256 的因数
    //    所以限制 head_dim 需要是 16/32/64/128/256，且额外要求大于等于32
    TORCH_CHECK(
        head_dim == 32 ||
        head_dim == 64 || head_dim == 128 || 
        head_dim == 256,
        "query head_dim must be in {16, 32, 64, 128, 256}"
    );
    TORCH_CHECK(
        block_table.size(0) == batch_size &&
        num_full_blocks.size(0) == batch_size &&
        out.size(0) == batch_size,
        "must have the same batch_size"
    );
    TORCH_CHECK(
        key_meta_cache.size(-1) == head_dim,
        "key_meta_cache must have the same head_dim with q"
    );

    const at::cuda::OptionalCUDAGuard device_guard(device_of(q));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    DISPATCH_HALF_AND_BF16(q.scalar_type(), "lserve_page_selector", [&] {
        using DataType = typename SATypeConverter<scalar_t>::Type;

        lserve_page_selector_head_dim_dispatcher<DataType>(
            reinterpret_cast<DataType*>(q.data_ptr()),
            reinterpret_cast<DataType*>(key_meta_cache.data_ptr()),
            block_table.data_ptr<int>(),
            num_full_blocks.data_ptr<int>(),
            reinterpret_cast<DataType*>(out.data_ptr()),
            batch_size,
            num_q_head,
            num_kv_head,
            head_dim,
            max_block_size,
            q.stride(0),
            q.stride(1)
        );
    });
}