#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_token_rotary_embedding(
    scalar_t* __restrict__ arr, const scalar_t* __restrict__ cos_ptr,
    const scalar_t* __restrict__ sin_ptr, int rot_offset, int embed_dim) {
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_token_dca_rotary_embedding(
  const scalar_t* __restrict__ arr,
  scalar_t* __restrict__ out,
  const scalar_t* __restrict__ q_cos_ptr,
  const scalar_t* __restrict__ q_sin_ptr,
  const scalar_t* __restrict__ q_succ_cos_ptr,
  const scalar_t* __restrict__ q_succ_sin_ptr,
  const scalar_t* __restrict__ q_inter_cos_ptr,
  const scalar_t* __restrict__ q_inter_sin_ptr,
  const scalar_t* __restrict__ q_succ_c_cos_ptr,
  const scalar_t* __restrict__ q_succ_c_sin_ptr,
  const scalar_t* __restrict__ q_inter_c_cos_ptr,
  const scalar_t* __restrict__ q_inter_c_sin_ptr,
  int rot_offset, int embed_dim,
  int split_stride
) {
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    int x_index, y_index, ox_index, oy_index;
    scalar_t cos, sin;

    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    ox_index = x_index;
    oy_index = y_index;
    cos = VLLM_LDG(q_cos_ptr + x_index);
    sin = VLLM_LDG(q_sin_ptr + x_index);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_succ_cos_ptr + x_index);
    sin = VLLM_LDG(q_succ_sin_ptr + x_index);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_inter_cos_ptr + x_index);
    sin = VLLM_LDG(q_inter_sin_ptr + x_index);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_succ_c_cos_ptr + x_index);
    sin = VLLM_LDG(q_succ_c_sin_ptr + x_index);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_inter_c_cos_ptr + x_index);
    sin = VLLM_LDG(q_inter_c_sin_ptr + x_index);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;
  } else {
    // GPT-J style rotary embedding.
    int x_index, y_index, ox_index, oy_index;
    scalar_t cos, sin;

    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    ox_index = x_index;
    oy_index = y_index;
    cos = VLLM_LDG(q_cos_ptr + x_index / 2);
    sin = VLLM_LDG(q_sin_ptr + x_index / 2);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_succ_cos_ptr + x_index / 2);
    sin = VLLM_LDG(q_succ_sin_ptr + x_index / 2);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_inter_cos_ptr + x_index / 2);
    sin = VLLM_LDG(q_inter_sin_ptr + x_index / 2);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_succ_c_cos_ptr + x_index / 2);
    sin = VLLM_LDG(q_succ_c_sin_ptr + x_index / 2);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;

    ox_index += split_stride;
    oy_index += split_stride;
    cos = VLLM_LDG(q_inter_c_cos_ptr + x_index / 2);
    sin = VLLM_LDG(q_inter_c_sin_ptr + x_index / 2);
    out[ox_index] = arr[x_index] * cos - arr[y_index] * sin;
    out[oy_index] = arr[y_index] * cos + arr[x_index] * sin;
  }
}

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
    scalar_t* __restrict__ query,  // [batch_size, seq_len, num_heads,
                                   // head_size] or [num_tokens, num_heads,
                                   // head_size]
    scalar_t* __restrict__ key,    // nullptr or
                                   // [batch_size, seq_len, num_kv_heads,
                                   // head_size] or [num_tokens, num_kv_heads,
                                   // head_size]
    const scalar_t* cache_ptr, const int head_size, const int num_heads,
    const int num_kv_heads, const int rot_dim, const int token_idx,
    const int64_t query_stride, const int64_t key_stride,
    const int64_t head_stride) {
  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head =
        token_idx * query_stride + head_idx * head_stride;
    const int rot_offset = i % embed_dim;
    apply_token_rotary_embedding<scalar_t, IS_NEOX>(
        query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }

  if (key != nullptr) {
    const int nk = num_kv_heads * embed_dim;
    for (int i = threadIdx.x; i < nk; i += blockDim.x) {
      const int head_idx = i / embed_dim;
      const int64_t token_head =
          token_idx * key_stride + head_idx * head_stride;
      const int rot_offset = i % embed_dim;
      apply_token_rotary_embedding<scalar_t, IS_NEOX>(
          key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
    }
  }
}

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_dca_rotary_embedding(
  const scalar_t* __restrict__ query, // [batch_size, seq_len, num_heads * head_size] or
                                // [num_tokens, num_heads * head_size] or
                                // [batch_size, seq_len, num_heads, head_size] or
                                // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,   // [batch_size, seq_len, num_kv_heads * head_size] or
                                // [num_tokens, num_kv_heads * head_size] or
                                // [batch_size, seq_len, num_kv_heads, head_size] or
                                // [num_tokens, num_kv_heads, head_size]
  scalar_t* __restrict__ qout,
  const scalar_t* q_cache_ptr,
  const scalar_t* q_succ_cache_ptr,
  const scalar_t* q_inter_cache_ptr,
  const scalar_t* q_succ_critical_cache_ptr,
  const scalar_t* q_inter_critical_cache_ptr,
  const int head_size, 
  const int num_heads, const int num_kv_heads, 
  const int rot_dim, 
  const int token_idx,
  const int64_t query_stride, const int64_t key_stride, const int64_t head_stride,
  const int64_t out_stride
) {
  const int embed_dim = rot_dim / 2;
  const scalar_t* q_cos_ptr = q_cache_ptr;
  const scalar_t* q_sin_ptr = q_cache_ptr + embed_dim;
  const scalar_t* q_succ_cos_ptr = q_succ_cache_ptr;
  const scalar_t* q_succ_sin_ptr = q_succ_cache_ptr + embed_dim;
  const scalar_t* q_inter_cos_ptr = q_inter_cache_ptr;
  const scalar_t* q_inter_sin_ptr = q_inter_cache_ptr + embed_dim;
  const scalar_t* q_succ_c_cos_ptr = q_succ_critical_cache_ptr;
  const scalar_t* q_succ_c_sin_ptr = q_succ_critical_cache_ptr + embed_dim;
  const scalar_t* q_inter_c_cos_ptr = q_inter_critical_cache_ptr;
  const scalar_t* q_inter_c_sin_ptr = q_inter_critical_cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    int head_idx = i / embed_dim;
    const int64_t token_head =
        token_idx * query_stride + head_idx * head_stride;
    const int64_t out_token_head =
        token_idx * out_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_token_dca_rotary_embedding<scalar_t, IS_NEOX>(
      query + token_head,
      // FIXIT
      qout + out_token_head,
      q_cos_ptr, q_sin_ptr,
      q_succ_cos_ptr, q_succ_sin_ptr,
      q_inter_cos_ptr, q_inter_sin_ptr,
      q_succ_c_cos_ptr, q_succ_c_sin_ptr,
      q_inter_c_cos_ptr, q_inter_c_sin_ptr,
      rot_offset, embed_dim,
      out_stride / 5);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head =
        token_idx * key_stride + head_idx * head_stride;
    const int rot_offset = i % embed_dim;
    apply_token_rotary_embedding<scalar_t, IS_NEOX>(
      key + token_head, q_cos_ptr, q_sin_ptr, rot_offset, embed_dim);
  }
}

template <typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
    const int64_t* __restrict__ positions,  // [batch_size, seq_len] or
                                            // [num_tokens]
    scalar_t* __restrict__ query,           // [batch_size, seq_len, num_heads,
                                   // head_size] or [num_tokens, num_heads,
                                   // head_size]
    scalar_t* __restrict__ key,  // nullptr or
                                 // [batch_size, seq_len, num_kv_heads,
                                 // head_size] or [num_tokens, num_kv_heads,
                                 // head_size]
    const scalar_t* __restrict__ cos_sin_cache,  // [max_position, 2, rot_dim //
                                                 // 2]
    const int rot_dim, const int64_t query_stride, const int64_t key_stride,
    const int64_t head_stride, const int num_heads, const int num_kv_heads,
    const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  apply_rotary_embedding<scalar_t, IS_NEOX>(
      query, key, cache_ptr, head_size, num_heads, num_kv_heads, rot_dim,
      token_idx, query_stride, key_stride, head_stride);
}

template <typename scalar_t, bool IS_NEOX>
__global__ void dca_rotary_embedding_kernel(
  const int64_t* __restrict__ positions,  // [batch_size, seq_len] or [num_tokens]
  const scalar_t* __restrict__ query,     // [batch_size, seq_len, num_heads * head_size] or
                                          // [num_tokens, num_heads * head_size] or
                                          // [batch_size, seq_len, num_heads, head_size] or
                                          // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,             // [batch_size, seq_len, num_kv_heads * head_size] or
                                          // [num_tokens, num_kv_heads * head_size] or
                                          // [batch_size, seq_len, num_kv_heads, head_size] or
                                          // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_q_cache,           // [chunk_len, 2, rot_dim // 2]
  const scalar_t* __restrict__ cos_sin_qc_cache,          // [chunk_len, 2, rot_dim // 2]
  const scalar_t* __restrict__ cos_sin_qc_no_clamp_cache, // [chunk_len, 2, rot_dim // 2]
  const scalar_t* __restrict__ cos_sin_q_inter_cache,     // [chunk_len, 2, rot_dim // 2]
  scalar_t* __restrict__ qout,
  const int rot_dim,
  const int64_t query_stride, const int64_t key_stride,
  const int64_t head_stride, const int64_t out_stride,
  const int num_heads, const int num_kv_heads,
  const int head_size,
  const int64_t chunk_len
) {
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* q_cache_ptr = cos_sin_q_cache + (pos % chunk_len) * rot_dim;
  // const scalar_t* k_cache_ptr = cos_sin_q_cache + (pos % chunk_len) * rot_dim;
  const scalar_t* q_succ_cache_ptr = cos_sin_qc_cache + (pos % chunk_len) * rot_dim;
  const scalar_t* q_inter_cache_ptr = cos_sin_qc_cache + (chunk_len - 1) * rot_dim;
  const scalar_t* q_succ_critical_cache_ptr = 
    cos_sin_qc_no_clamp_cache + (pos % chunk_len) * rot_dim;
  const scalar_t* q_inter_critical_cache_ptr = 
    cos_sin_q_inter_cache + (pos % chunk_len) * rot_dim;
  
  apply_dca_rotary_embedding<scalar_t, IS_NEOX>(
    query, key, 
    qout,
    q_cache_ptr,
    // k_cache_ptr,
    q_succ_cache_ptr,
    q_inter_cache_ptr,
    q_succ_critical_cache_ptr,
    q_inter_critical_cache_ptr,
    head_size, 
    num_heads, num_kv_heads, 
    rot_dim,
    token_idx, 
    query_stride, key_stride, head_stride,
    out_stride
  );
}

template <typename scalar_t, bool IS_NEOX>
__global__ void batched_rotary_embedding_kernel(
    const int64_t* __restrict__ positions,  // [batch_size, seq_len] or
                                            // [num_tokens]
    scalar_t* __restrict__ query,           // [batch_size, seq_len, num_heads,
                                   // head_size] or [num_tokens, num_heads,
                                   // head_size]
    scalar_t* __restrict__ key,  // nullptr or
                                 // [batch_size, seq_len, num_kv_heads,
                                 // head_size] or [num_tokens, num_kv_heads,
                                 // head_size]
    const scalar_t* __restrict__ cos_sin_cache,  // [max_position, 2, rot_dim //
                                                 // 2]
    const int64_t* __restrict__ cos_sin_cache_offsets,  // [batch_size, seq_len]
    const int rot_dim, const int64_t query_stride, const int64_t key_stride,
    const int64_t head_stride, const int num_heads, const int num_kv_heads,
    const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  int64_t cos_sin_cache_offset = cos_sin_cache_offsets[token_idx];
  const scalar_t* cache_ptr =
      cos_sin_cache + (cos_sin_cache_offset + pos) * rot_dim;

  apply_rotary_embedding<scalar_t, IS_NEOX>(
      query, key, cache_ptr, head_size, num_heads, num_kv_heads, rot_dim,
      token_idx, query_stride, key_stride, head_stride);
}

}  // namespace vllm

void dca_rotary_embedding(
  torch::Tensor& positions,                   // [batch_size, seq_len] or [num_tokens]
  torch::Tensor& query,                       // [batch_size, seq_len, num_heads * head_size] or
                                              // [num_tokens, num_heads * head_size] or
                                              // [batch_size, seq_len, num_heads, head_size] or
                                              // [num_tokens, num_heads, head_size]
  torch::Tensor& key,                         // [batch_size, seq_len, num_kv_heads * head_size] or
                                              // [num_tokens, num_kv_heads * head_size] or
                                              // [batch_size, seq_len, num_kv_heads, head_size] or
                                              // [num_tokens, num_kv_heads, head_size]
  int64_t head_size,
  torch::Tensor& cos_sin_q_cache,             // [chunk_len, rot_dim]
  torch::Tensor& cos_sin_qc_cache,            // [chunk_len, rot_dim]
  torch::Tensor& cos_sin_qc_no_clamp_cache,   // [chunk_len, rot_dim]
  torch::Tensor& cos_sin_q_inter_cache,       // [chunk_len, rot_dim]
  torch::Tensor& out,
  int64_t chunk_len,
  bool is_neox
) {
  // query & key is not contiguous because of torch.split
  TORCH_CHECK(
    positions.is_contiguous() &&
    cos_sin_q_cache.is_contiguous() &&
    cos_sin_qc_cache.is_contiguous() &&
    cos_sin_qc_no_clamp_cache.is_contiguous() &&
    cos_sin_q_inter_cache.is_contiguous() &&
    out.is_contiguous(),
    "all tensor must be contiguous"
  );

  int64_t num_tokens = positions.numel();
  int positions_ndim = positions.dim();

  TORCH_CHECK(
      positions_ndim == 1 || positions_ndim == 2,
      "positions must have shape [num_tokens] or [batch_size, seq_len]");
  if (positions_ndim == 1) {
    TORCH_CHECK(query.size(0) == positions.size(0) &&
                out.size(0) == positions.size(0) &&
                key.size(0) == positions.size(0),
                "query, key, out and positions must have the same number of tokens");
  }
  if (positions_ndim == 2) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) &&
        key.size(0) == positions.size(0) &&
        out.size(0) == positions.size(0) &&
        query.size(1) == positions.size(1) &&
        key.size(1) == positions.size(1) &&
        out.size(1) == positions.size(1),
        "query, key, out and positions must have the same batch_size and seq_len");
  }
  
  int query_hidden_size = query.numel() / num_tokens;
  int key_hidden_size = key.numel() / num_tokens;
  TORCH_CHECK(query_hidden_size % head_size == 0);
  TORCH_CHECK(key_hidden_size % head_size == 0);
  TORCH_CHECK(out.numel() / num_tokens == query_hidden_size * 5);

  int num_heads = query_hidden_size / head_size;
  int num_kv_heads = key_hidden_size / head_size;
  TORCH_CHECK(num_heads % num_kv_heads == 0);

  int rot_dim = cos_sin_q_cache.size(1);
  TORCH_CHECK(
    cos_sin_qc_cache.size(1) == rot_dim &&
    cos_sin_qc_no_clamp_cache.size(1) == rot_dim &&
    cos_sin_q_inter_cache.size(1) == rot_dim,
    "cos sin cache must have the same rot_dim"
  );
  TORCH_CHECK(rot_dim == head_size, "rot_dim and head_size must be the same");

  int seq_dim_idx = positions_ndim - 1;
  int64_t query_stride = query.stride(seq_dim_idx);
  int64_t key_stride = key.stride(seq_dim_idx);
  int64_t out_stride = out.stride(seq_dim_idx);
  TORCH_CHECK(out_stride % 5 == 0);

  int query_ndim = query.dim();
  int64_t head_stride =
      (query_ndim == positions_ndim + 2) ? query.stride(-2) : head_size;
  
  dim3 grid(num_tokens);
  dim3 block(std::min<int64_t>(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(query.scalar_type(), "dca_rotary_embedding", [&] {
    if (is_neox) {
      vllm::dca_rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_q_cache.data_ptr<scalar_t>(),
          cos_sin_qc_cache.data_ptr<scalar_t>(),
          cos_sin_qc_no_clamp_cache.data_ptr<scalar_t>(),
          cos_sin_q_inter_cache.data_ptr<scalar_t>(),
          out.data_ptr<scalar_t>(),
          rot_dim,
          query_stride, key_stride,
          head_stride, out_stride,
          num_heads, num_kv_heads,
          head_size,
          chunk_len);
    } else {
      vllm::dca_rotary_embedding_kernel<scalar_t, false>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(),
              query.data_ptr<scalar_t>(),
              key.data_ptr<scalar_t>(),
              cos_sin_q_cache.data_ptr<scalar_t>(),
              cos_sin_qc_cache.data_ptr<scalar_t>(),
              cos_sin_qc_no_clamp_cache.data_ptr<scalar_t>(),
              cos_sin_q_inter_cache.data_ptr<scalar_t>(),
              out.data_ptr<scalar_t>(),
              rot_dim,
              query_stride, key_stride,
              head_stride, out_stride,
              num_heads, num_kv_heads,
              head_size,
              chunk_len);
    }
  });
}

void rotary_embedding(
    torch::Tensor& positions,  // [batch_size, seq_len] or [num_tokens]
    torch::Tensor& query,  // [batch_size, seq_len, num_heads * head_size] or
                           // [num_tokens, num_heads * head_size] or
                           // [batch_size, seq_len, num_heads, head_size] or
                           // [num_tokens, num_heads, head_size]
    std::optional<torch::Tensor> key,
    // null or
    // [batch_size, seq_len, num_kv_heads * head_size] or
    // [num_tokens, num_kv_heads * head_size] or
    // [batch_size, seq_len, num_heads, head_size] or
    // [num_tokens, num_heads, head_size]
    int64_t head_size,
    torch::Tensor& cos_sin_cache,  // [max_position, rot_dim]
    bool is_neox) {
  // num_tokens = batch_size * seq_len
  int64_t num_tokens = positions.numel();
  int positions_ndim = positions.dim();

  // Make sure num_tokens dim is consistent across positions, query, and key
  TORCH_CHECK(
      positions_ndim == 1 || positions_ndim == 2,
      "positions must have shape [num_tokens] or [batch_size, seq_len]");
  if (positions_ndim == 1) {
    TORCH_CHECK(query.size(0) == positions.size(0) &&
                    (!key.has_value() || key->size(0) == positions.size(0)),
                "query, key and positions must have the same number of tokens");
  }
  if (positions_ndim == 2) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) &&
            (!key.has_value() || key->size(0) == positions.size(0)) &&
            query.size(1) == positions.size(1) &&
            (!key.has_value() || key->size(1) == positions.size(1)),
        "query, key and positions must have the same batch_size and seq_len");
  }

  // Make sure head_size is valid for query and key
  // hidden_size = num_heads * head_size
  int query_hidden_size = query.numel() / num_tokens;
  int key_hidden_size = key.has_value() ? key->numel() / num_tokens : 0;
  TORCH_CHECK(query_hidden_size % head_size == 0);
  TORCH_CHECK(key_hidden_size % head_size == 0);

  // Make sure query and key have consistent number of heads
  int num_heads = query_hidden_size / head_size;
  int num_kv_heads = key.has_value() ? key_hidden_size / head_size : num_heads;
  TORCH_CHECK(num_heads % num_kv_heads == 0);

  int rot_dim = cos_sin_cache.size(1);
  int seq_dim_idx = positions_ndim - 1;
  int64_t query_stride = query.stride(seq_dim_idx);
  int64_t key_stride = key.has_value() ? key->stride(seq_dim_idx) : 0;
  // Determine head stride: for [*, heads, head_size] use stride of last dim;
  // for flat [*, heads*head_size], heads blocks are contiguous of size
  // head_size
  int query_ndim = query.dim();
  int64_t head_stride =
      (query_ndim == positions_ndim + 2) ? query.stride(-2) : head_size;

  dim3 grid(num_tokens);
  dim3 block(std::min<int64_t>(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding", [&] {
    if (is_neox) {
      vllm::rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
          key.has_value() ? key->data_ptr<scalar_t>() : nullptr,
          cos_sin_cache.data_ptr<scalar_t>(), rot_dim, query_stride, key_stride,
          head_stride, num_heads, num_kv_heads, head_size);
    } else {
      vllm::rotary_embedding_kernel<scalar_t, false>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
              key.has_value() ? key->data_ptr<scalar_t>() : nullptr,
              cos_sin_cache.data_ptr<scalar_t>(), rot_dim, query_stride,
              key_stride, head_stride, num_heads, num_kv_heads, head_size);
    }
  });
}

/*
Batched version of rotary embedding, pack multiple LoRAs together
and process in batched manner.
*/
void batched_rotary_embedding(
    torch::Tensor& positions,  // [batch_size, seq_len] or [num_tokens]
    torch::Tensor& query,  // [batch_size, seq_len, num_heads * head_size] or
                           // [num_tokens, num_heads * head_size] or
                           // [batch_size, seq_len, num_heads, head_size] or
                           // [num_tokens, num_heads, head_size]
    std::optional<torch::Tensor>
        key,  // null or
              // [batch_size, seq_len, num_kv_heads * head_size] or
              // [num_tokens, num_kv_heads * head_size] or
              // [batch_size, seq_len, num_heads, head_size] or
              // [num_tokens, num_heads, head_size]
    int64_t head_size,
    torch::Tensor& cos_sin_cache,  // [max_position, rot_dim]
    bool is_neox, int64_t rot_dim,
    torch::Tensor& cos_sin_cache_offsets  // [num_tokens] or [batch_size]
) {
  // num_tokens = batch_size * seq_len
  int64_t num_tokens = cos_sin_cache_offsets.size(0);
  TORCH_CHECK(
      positions.size(0) == num_tokens || positions.numel() == num_tokens,
      "positions must have the same num_tokens or batch_size as "
      "cos_sin_cache_offsets");

  int positions_ndim = positions.dim();
  // Make sure num_tokens dim is consistent across positions, query, and key
  TORCH_CHECK(
      positions_ndim == 1 || positions_ndim == 2,
      "positions must have shape [num_tokens] or [batch_size, seq_len]");
  if (positions_ndim == 1) {
    TORCH_CHECK(query.size(0) == positions.size(0) &&
                    (!key.has_value() || key->size(0) == positions.size(0)),
                "query, key and positions must have the same number of tokens");
  }
  if (positions_ndim == 2) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) &&
            (!key.has_value() || key->size(0) == positions.size(0)) &&
            query.size(1) == positions.size(1) &&
            (!key.has_value() || key->size(1) == positions.size(1)),
        "query, key and positions must have the same batch_size and seq_len");
  }

  // Make sure head_size is valid for query and key
  int query_hidden_size = query.numel() / num_tokens;
  int key_hidden_size = key.has_value() ? key->numel() / num_tokens : 0;
  TORCH_CHECK(query_hidden_size % head_size == 0);
  TORCH_CHECK(key_hidden_size % head_size == 0);

  // Make sure query and key have concistent number of heads
  int num_heads = query_hidden_size / head_size;
  int num_kv_heads = key.has_value() ? key_hidden_size / head_size : num_heads;
  TORCH_CHECK(num_heads % num_kv_heads == 0);

  int seq_dim_idx = positions_ndim - 1;
  int64_t query_stride = query.stride(seq_dim_idx);
  int64_t key_stride = key.has_value() ? key->stride(seq_dim_idx) : 0;
  // Determine head stride: for [*, heads, head_size] use stride of last dim;
  // for flat [*, heads*head_size], heads blocks are contiguous of size
  // head_size
  int query_ndim = query.dim();
  int64_t head_stride =
      (query_ndim == positions_ndim + 2) ? query.stride(-2) : head_size;

  dim3 grid(num_tokens);
  dim3 block(std::min<int64_t>(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding", [&] {
    if (is_neox) {
      vllm::batched_rotary_embedding_kernel<scalar_t, true>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
              key.has_value() ? key->data_ptr<scalar_t>() : nullptr,
              cos_sin_cache.data_ptr<scalar_t>(),
              cos_sin_cache_offsets.data_ptr<int64_t>(), rot_dim, query_stride,
              key_stride, head_stride, num_heads, num_kv_heads, head_size);
    } else {
      vllm::batched_rotary_embedding_kernel<scalar_t, false>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
              key.has_value() ? key->data_ptr<scalar_t>() : nullptr,
              cos_sin_cache.data_ptr<scalar_t>(),
              cos_sin_cache_offsets.data_ptr<int64_t>(), rot_dim, query_stride,
              key_stride, head_stride, num_heads, num_kv_heads, head_size);
    }
  });
}
